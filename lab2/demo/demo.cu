#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "lib/scmp.cu"

const int n = 5;
const size_t size = sizeof(float) * n;
//for host
float *vector_a_x, *vector_a_y, *vector_b_x, *vector_b_y, *result;
//for device
float *d_vector_a_x, *d_vector_a_y, *d_vector_b_x, *d_vector_b_y, *d_result;

void allocateMemory();
void loadValues();
void calcOnCPU();
void calcOnGPU();
void clear();

int main()
{
    allocateMemory();
    loadValues();
    calcOnCPU();
    calcOnGPU();
    clear();
}

void allocateMemory()
{
    vector_a_x = (float*)malloc(size);
    vector_a_y = (float*)malloc(size);
    vector_b_x = (float*)malloc(size);
    vector_b_y = (float*)malloc(size);
    result = (float*)malloc(size);
    hipMalloc(&d_vector_a_x,size);
    hipMalloc(&d_vector_a_y,size);
    hipMalloc(&d_vector_b_x,size);
    hipMalloc(&d_vector_b_y,size);
    hipMalloc(&d_result,size);
}

void loadValues()
{
    srand(0);
    for (size_t i = 0; i < n; i++)
    {
        vector_a_x[i] = (float(rand()+1) / float(rand()+1));
        vector_a_y[i] = (float(rand()+1) / float(rand()+1));
        vector_b_x[i] = (float(rand()+1) / float(rand()+1));
        vector_b_y[i] = (float(rand()+1) / float(rand()+1));
        std::cout << vector_a_x[i] << " " << vector_a_y[i] << " " << vector_b_x[i] << " " << vector_b_y[i] << std::endl;
    }
    std::cout << std::endl;
}

void calcOnCPU()
{
    scmpOnCPU(vector_a_x, vector_a_y, vector_b_x, vector_b_y, result, n);

    for (size_t i = 0; i < n; i++)
    {
        std::cout << result[i] << std::endl;
    }
    std::cout << std::endl;
}

void calcOnGPU()
{
    hipMemcpy(d_vector_a_x, vector_a_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_a_y, vector_a_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b_x, vector_b_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b_y, vector_b_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, size, hipMemcpyHostToDevice);

    int bl, th;
    if (n > 1024)
    {
        bl = (n / 1024) + 1;
        th = 1024;
    }
    else
    {
        th = n;
        bl = 1;
    }

    scmpOnGPU <<<bl, th >>> (d_vector_a_x, d_vector_a_y, d_vector_b_x, d_vector_b_y, d_result);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < n; i++)
    {
        std::cout << result[i] << std::endl;
    }
    std::cout << std::endl;
}

void clear()
{
    hipFree(d_vector_a_x);
    hipFree(d_vector_a_y);
    hipFree(d_vector_b_x);
    hipFree(d_vector_b_y);
    hipFree(d_result);
    delete[] vector_a_x;
    delete[] vector_a_y;
    delete[] vector_b_x;
    delete[] vector_b_y;
    delete[] result;
}