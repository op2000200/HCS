#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void scmpOnGPU(float* vector_a, float* vector_b, float* result)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    result[i] = vector_a[i] * vector_b[i];
}

__host__ void scmpOnCPU(const float* vector_a, const float* vector_b, int size, float* result, int n)
{
    *result = 0;
    for (int i = 0; i < n; i++)
    {
        *result += vector_a[i] +vector_b[i];
    }
}

float calcOnGpu(torch::Tensor vec1, torch::Tensor vec2)
{
    int size = vec1.size(0);
    float *d_vector_a, *d_vector_b, *d_result;
    float *res = new float[size];
    hipMalloc(&d_vector_a, sizeof(float) * size);
    hipMalloc(&d_vector_b, sizeof(float) * size);
    hipMalloc(&d_result, sizeof(float) * size);

    hipMemcpy(d_vector_a, vec1.data_ptr<float>(), sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b, vec2.data_ptr<float>(), sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, res, sizeof(float) * size, hipMemcpyHostToDevice);

    int bl, th;
    if (size > 1024)
    {
        bl = (size / 1024) + 1;
        th = 1024;
    }
    else
    {
        th = size;
        bl = 1;
    }
    
    scmpOnGPU <<<bl, th >>> (d_vector_a, d_vector_b, d_result);

    hipDeviceSynchronize();

    // Копируем результат обратно на хост
    hipMemcpy(res, d_result, sizeof(float) * size, hipMemcpyDeviceToHost);

    // Освобождаем память на устройстве
    hipFree(d_vector_a);
    hipFree(d_vector_b);
    hipFree(d_result);
    float sum = 0;
    for (size_t i = 0; i < size; i++)
    {
        sum += res[i];
    }
    
    return sum;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("dot_gpu", &calcOnGpu);
}