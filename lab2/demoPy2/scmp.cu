#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void scmpOnGPU(const float* vector_a_x, const float* vector_a_y, const float* vector_b_x, const float* vector_b_y, float* result)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    result[i] = vector_a_x[i] * vector_b_x[i] + vector_a_y[i] * vector_b_y[i];
}

__host__ void scmpOnCPU(const float* vector_a_x, const float* vector_a_y, const float* vector_b_x, const float* vector_b_y, float* result, const int n)
{
    for (size_t i = 0; i < n; i++)
    {
        result[i] = vector_a_x[i] * vector_b_x[i] + vector_a_y[i] * vector_b_y[i];
    }
}


torch::Tensor test_dummy(torch::Tensor vec1, torch::Tensor vec2, torch::Tensor vec3, torch::Tensor vec4)
{
    torch::Tensor res = torch::empty(1);
    return res;
}


torch::Tensor calcOnCpu(torch::Tensor vec1, torch::Tensor vec2, torch::Tensor vec3, torch::Tensor vec4)
{
    int size = vec1.size(0);
    float *vector_a_x, *vector_a_y, *vector_b_x, *vector_b_y, *result;
    vector_a_x = (float*)malloc(sizeof(float) * size);
    vector_a_x = vec1.data<float>();
    vector_a_y = (float*)malloc(sizeof(float) * size);
    vector_a_y = vec2.data<float>();
    vector_b_x = (float*)malloc(sizeof(float) * size);
    vector_b_x = vec3.data<float>();
    vector_b_y = (float*)malloc(sizeof(float) * size);
    vector_b_y = vec4.data<float>();
    result = (float*)malloc(sizeof(float) * size);
    scmpOnCPU(vector_a_x, vector_a_y, vector_b_x, vector_b_y, result, size);
    
    torch::Tensor res = torch::empty(size);
    for (int i = 0; i < size; i++)
    {
      res[i] = result[i];
      std::cout << result[i] << " " << res[i] << std::endl;
    }

    delete[] vector_a_x;
    delete[] vector_a_y;
    delete[] vector_b_x;
    delete[] vector_b_y;
    delete[] result;

    return res;
}

torch::Tensor calcOnGpu(torch::Tensor vec1, torch::Tensor vec2, torch::Tensor vec3, torch::Tensor vec4)
{
    int size = vec1.size(0);
    float *vector_a_x, *vector_a_y, *vector_b_x, *vector_b_y, *result;
    float *d_vector_a_x, *d_vector_a_y, *d_vector_b_x, *d_vector_b_y, *d_result;
    vector_a_x = (float*)malloc(sizeof(float) * size);
    vector_a_x = vec1.data<float>();
    vector_a_y = (float*)malloc(sizeof(float) * size);
    vector_a_y = vec2.data<float>();
    vector_b_x = (float*)malloc(sizeof(float) * size);
    vector_b_x = vec3.data<float>();
    vector_b_y = (float*)malloc(sizeof(float) * size);
    vector_b_y = vec4.data<float>();
    result = (float*)malloc(sizeof(float) * size);
    hipMalloc(&d_vector_a_x,sizeof(float) * size);
    hipMalloc(&d_vector_a_y,sizeof(float) * size);
    hipMalloc(&d_vector_b_x,sizeof(float) * size);
    hipMalloc(&d_vector_b_y,sizeof(float) * size);
    hipMalloc(&d_result,sizeof(float) * size);
    hipMemcpy(d_vector_a_x, vector_a_x, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_a_y, vector_a_y, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b_x, vector_b_x, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b_y, vector_b_y, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, sizeof(float) * size, hipMemcpyHostToDevice);
    
    int bl, th;
    if (size > 1024)
    {
        bl = (size / 1024) + 1;
        th = 1024;
    }
    else
    {
        th = size;
        bl = 1;
    }
    
    scmpOnGPU <<<bl, th >>> (d_vector_a_x, d_vector_a_y, d_vector_b_x, d_vector_b_y, d_result);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, sizeof(float) * size, hipMemcpyDeviceToHost);
    
    torch::Tensor res = torch::empty(size);
    for (int i = 0; i < size; i++)
    {
      res[i] = result[i];
      std::cout << result[i] << " " << res[i] << std::endl;
    }

    hipFree(d_vector_a_x);
    hipFree(d_vector_a_y);
    hipFree(d_vector_b_x);
    hipFree(d_vector_b_y);
    hipFree(d_result);
    delete[] vector_a_x;
    delete[] vector_a_y;
    delete[] vector_b_x;
    delete[] vector_b_y;
    delete[] result;

    return res;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("cpu", &calcOnCpu);
    m.def("gpu", &calcOnGpu);
    m.def("test", &test_dummy);
