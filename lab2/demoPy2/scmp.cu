#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void scmpOnGPU(float* vector_a, float* vector_b, int size, float* result)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    *result = vector_a[i] * vector_b[i];
}

__host__ void scmpOnCPU(const float** vector_a, const float** vector_b, int size, float* result, int n)
{
    for (int i = 0; i < n; i++)
    {
        float buffer = 0;
        for (int j = 0; j < size; j++)
        {
            buffer += vector_a[i][j] * vector_b[i][j];
        }
        result[i] = buffer;
    }
}

float calcOnGpu(torch::Tensor vec1, torch::Tensor vec2)
{
    int size = vec1.size(0);
    float *d_vector_a, *d_vector_b, *d_result;
    float buf = 0.f;
    hipMalloc(&d_vector_a, sizeof(float) * size);
    hipMalloc(&d_vector_b, sizeof(float) * size);
    hipMalloc(&d_result, sizeof(float));

    hipMemcpy(d_vector_a, vec1.data_ptr<float>(), sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_b, vec1.data_ptr<float>(), sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, &buf, sizeof(float), hipMemcpyHostToDevice);

    *d_result = 0;

    int bl, th;
    if (size > 1024)
    {
        bl = (size / 1024) + 1;
        th = 1024;
    }
    else
    {
        th = size;
        bl = 1;
    }
    
    scmpOnGPU <<<bl, th >>> (d_vector_a, d_vector_b, size, d_result);

    hipDeviceSynchronize();
    
    float res = *d_result;
    std::cout << res << std::endl;
    return res;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("dot_gpu", &calcOnGpu);
}